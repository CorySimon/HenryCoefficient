#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string>
#include <fstream>
#include <sstream>
#include <map>
#include <hiprand/hiprand_kernel.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define NUMBLOCKS 3
#define NUMTHREADS 256

// data for atom of crystal structure
//    Unit cell of crystal structure can then be stored 
//    as pointer array of StructureAtom's
struct StructureAtom {
    // Cartesian position, units: A
    double x;
    double y;
    double z;
    // Lennard-Jones epsilon parameter with adsorbate
    double epsilon;  // units: K
    // Lennard-Jones sigma parameter with adsorbate
    double sigma;  // units: A
};

// temperature, Kelvin
const double T = 298.0; 

// Universal gas constant, m3 - Pa / (K - mol)
const double R = 8.314; 

// Number of times to call GPU kernel
int ninsertions = 100000 * 256 ;
int ncycles = floor(ninsertions / (NUMTHREADS * NUMBLOCKS));

// Compute the Boltzmann factor of methane at point (x, y, z) inside structure
//   Loop over all atoms of unit cell of crystal structure
//   Find nearest image to methane at point (x, y, z) for application of periodic boundary conditions
//   Compute energy contribution due to this atom via the Lennard-Jones potential
__device__ double ComputeBoltzmannFactorAtPoint(double x, double y, double z,
                                                const StructureAtom * __restrict__ structureatoms,
                                                double natoms,
                                                double L) {
    // (x, y, z) : Cartesian coords of methane molecule
    // structureatoms : pointer array storing info on unit cell of crystal structure
    // natoms : number of atoms in crystal structure
    // L : box length
    double E = 0.0;
    
    // loop over atoms in crystal structure
    for (int i = 0; i < natoms; i++) {
        //  Compute distance from (x, y, z) to this atom

        // compute distances in each coordinate
        double dx = x - structureatoms[i].x;
        double dy = y - structureatoms[i].y;
        double dz = z - structureatoms[i].z;
        
        // apply nearest image convention for periodic boundary conditions
        if (dx > L / 2.0)
            dx = dx - L;
        if (dy > L / 2.0)
            dy = dy - L;
        if (dz > L / 2.0)
            dz = dz - L;
        if (dx <= -L / 2.0)
            dx = dx + L;
        if (dy <= -L / 2.0)
            dy = dy + L;
        if (dy <= -L / 2.0)
            dy = dy + L;

        // distance
        double rinv = rsqrt(dx*dx + dy*dy + dz*dz);

        // Compute contribution to energy of adsorbate at (x, y, z) due to this atom
        // Lennard-Jones potential (not efficient, but for clarity)
        E += 4.0 * structureatoms[i].epsilon * (pow(structureatoms[i].sigma * rinv, 12) - pow(structureatoms[i].sigma * rinv, 6));
    }
    return exp(-E / (R * T));  // return Boltzmann factor
}

// Inserts a methane molecule at a random position inside the structure
// Calls function to compute Boltzmann factor at this point
// Stores Boltzmann factor computed at this thread in deviceBoltzmannFactors
__global__ void PerformInsertions(hiprandStateMtgp32 *state, 
                                  double * boltzmannFactors, 
                                  const StructureAtom * __restrict__ structureatoms, 
                                  int natoms, double L) {
    // state : random number generator
    // boltzmannFactors : pointer array in which to store computed Boltzmann factors
    // structureatoms : pointer array storing info on unit cell of crystal structure
    // natoms : number of atoms in crystal structure
    // L : box length
    int id = threadIdx.x + blockIdx.x * NUMTHREADS;  // thread ID
    
    // Generate random position inside the cubic unit cell of the structure
    double x = L * hiprand_uniform_double(&state[blockIdx.x]);
    double y = L * hiprand_uniform_double(&state[blockIdx.x]);
    double z = L * hiprand_uniform_double(&state[blockIdx.x]);

    // Compute Boltzmann factor, store in boltzmannFactors
    boltzmannFactors[id] = ComputeBoltzmannFactorAtPoint(x, y, z, structureatoms, natoms, L);
}

int main() {
    //
    // Energetic model for interactions of methane molecule with atoms of framework
    //    pairwise Lennard-Jones potentials
    //
    // Epsilon parameters for Lennard-Jones potential (K)
    std::map<std::string, double> epsilons;
    epsilons["Zn"] = 96.152688;
    epsilons["O"] = 66.884614;
    epsilons["C"] = 88.480032;
    epsilons["H"] = 57.276566;
    
    // Sigma parameters for Lennard-Jones potential (A)
    std::map<std::string, double> sigmas;
    sigmas["Zn"] = 3.095775;
    sigmas["O"] = 3.424075;
    sigmas["C"] = 3.580425;
    sigmas["H"] = 3.150565;

    //
    // Import unit cell of nanoporous material IRMOF-1
    //
    StructureAtom *structureatoms;  // store data in pointer array here
    // open crystal structure file
    std::ifstream materialfile("IRMOF-1.cssr");
    if (materialfile.fail()) {
        printf("IRMOF-1.cssr failed to import.\n");
        exit(EXIT_FAILURE);
    }

    // read cubic box dimensions
    std::string line;
    getline(materialfile, line);
    std::istringstream istream(line);

    double L;
    istream >> L;   
    printf("L = %f\n", L);

    // waste line
    getline(materialfile, line);
    
    // get number of atoms
    getline(materialfile, line);
    int natoms;  // number of atoms
    istream.str(line);
    istream.clear();
    istream >> natoms;
    printf("%d atoms\n", natoms);
    
    // waste line
    getline(materialfile, line);

    // Allocate space for material atoms and epsilons/sigmas on both host and device
    //   using unified memory
    CUDA_CALL(hipMallocManaged(&structureatoms, natoms * sizeof(StructureAtom)));

    // read atom coordinates
    for (int i = 0; i < natoms; i++) {
        getline(materialfile, line);
        istream.str(line);
        istream.clear();

        int atomno;
        double xf, yf, zf;  // fractional coordintes
        std::string element;

        istream >> atomno >> element >> xf >> yf >> zf;
        // load structureatoms
        structureatoms[i].x = L * xf;
        structureatoms[i].y = L * yf;
        structureatoms[i].z = L * zf;

        structureatoms[i].epsilon = epsilons[element];
        structureatoms[i].sigma = sigmas[element];

//        printf("%d. %s, (%f, %f, %f), eps = %f, sig = %f\n", 
//            atomno, element.c_str(), 
//            structureatoms[i].x, structureatoms[i].y, structureatoms[i].z,
//            structureatoms[i].epsilon,
//            structureatoms[i].sigma);
    }
    
    //
    // Allocate space for storing Botlzmann factors computed on each thread using unified memory
    //
    double * boltzmannFactors;
    CUDA_CALL(hipMallocManaged(&boltzmannFactors, NUMBLOCKS * NUMTHREADS, sizeof(double)));
    
    //
    // Set up random number generator on device
    //
    hiprandStateMtgp32 *devMTGPStates;
    mtgp32_kernel_params *devKernelParams;

    // Allocate space for prng states on device. One per block
    CUDA_CALL(hipMalloc((void **) &devMTGPStates, NUMBLOCKS * sizeof(hiprandStateMtgp32))); 
    
    // Setup MTGP prng states
    // Allocate space for MTGP kernel parameters
    CUDA_CALL(hipMalloc((void**) &devKernelParams, sizeof(mtgp32_kernel_params))); 
    
    // Reformat from predefined parameter sets to kernel format,
    // and copy kernel parameters to device memory
    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams)); 
    
    // Initialize one state per thread block
    CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, NUMBLOCKS, 1234)); 
    // State setup is complete
    
    //
    //  Compute the Henry coefficient in parallel
    //  KH = < e^{-E/(kB * T)} > / (R * T)
    //  Brackets denote average over space
    //
    double KH = 0.0;  // will be Henry coefficient
    for (int cycle = 0; cycle < ncycles; cycle++) {
        //  Perform Monte Carlo insertions in parallel on the GPU.
        PerformInsertions<<<NUMBLOCKS, NUMTHREADS>>>(devMTGPStates, boltzmannFactors, structureatoms, natoms, L);
        hipDeviceSynchronize();

        // Compute Henry coefficient from the sampled Boltzmann factors
        for(int i = 0; i < NUMBLOCKS * NUMTHREADS; i++) {
            KH += boltzmannFactors[i];
        }
    }
    // take averageBoltzmann constant
    KH = KH / (NUMBLOCKS * NUMTHREADS * ncycles);
    // at this point KH = < e^{-E/(kB/T)} >
    KH = KH / (R * T);
    printf("Henry constant = %e mol/(m3 - Pa)\n", KH);
    printf("Number of actual insertions: %d\n", NUMBLOCKS * NUMTHREADS * ncycles);
    printf("Number of times we called the GPU kernel: %d\n", ncycles);
    
    // Clean-up
    CUDA_CALL(hipFree(devMTGPStates));
    CUDA_CALL(hipFree(structureatoms));
    CUDA_CALL(hipFree(boltzmannFactors));
    return EXIT_SUCCESS;
}
